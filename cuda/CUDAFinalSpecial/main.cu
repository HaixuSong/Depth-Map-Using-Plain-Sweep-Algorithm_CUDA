#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>


struct planeParam {
    int from;
    int to;
    int numbers;
};

#define RESOLUTIONX 3072
#define RESOLUTIONY 2048

// constant memory declaration
__constant__ float pm3_d[12];
__constant__ float pm4_d[12];
__constant__ float pm5_d[12];
__constant__ float pm6_d[12];
__constant__ float pm7_d[12];

__constant__ float pi5_d[12];


void checkCUDAError(hipError_t e) {
    if (e == 0) return;
    printf("\nError: %s\n", hipGetErrorName(e));
    printf("%s\n", hipGetErrorString(e));
    exit(0);
}


int UI(int argc, char* argv[], struct planeParam* plp) {

	//input -h for help
	if (argc == 2 && (strcmp(argv[1], "--help") == 0 || strcmp(argv[1], "-h") == 0)) {
		printf("CUDA Version Plane Sweep Alogrithm\n");
		printf("\nUsage: psalgo [OPTION]...\n");
		printf("\nOptions:\n");
		printf("%5s, %-10s %-50s\n", "-h", "--help", "Show helping information.");
		printf("%5s, %-10s %-50s\n", "-r", "--range", "Followed by 3 integers as plane range and numbers.");
		printf("\nExplaining:\n");
		printf("Use , as seperator:\n");
		printf("  Use , as sepeartor in one parameter. For example, -r 4,9,60\n");
		printf("What --range followed by\n");
		printf("  Three integers. First two representing range, the third one representing how many planes."
			"Range could be either greater first or less first. We recognize it automatically. \n");
		printf("\nExamples:\n");
		printf("psalgo -h\n");
		printf("  Shows the helping information.\n");
		printf("psalgo -r 4,9,60\n");
		printf("  Planes are ranged from 4 to 9, 60 planes in total.\n");
		return 1;
	}

	// input range
	if (argc == 3 && (strcmp(argv[1], "-r") == 0 || strcmp(argv[1], "--range") == 0)) {
		// processing -r or --range
		char* pch;
		pch = strtok(argv[2], ",");
		if (pch == NULL) {
			printf("Invalid range input. Please check your command or use -h for help.\n");
			return 1;
		}
		plp->from = atoi(pch);
		pch = strtok(NULL, ",");
		if (pch == NULL) {
			printf("Invalid range input. Please check your command or use -h for help.\n");
			return 1;
		}
		plp->to = atoi(pch);
		pch = strtok(NULL, ",");
		if (pch == NULL) {
			printf("Invalid range input. Please check your command or use -h for help.\n");
			return 1;
		}
		plp->numbers = atoi(pch);

		// make plp.from > plp.to
		if (plp->from < plp->to) {
			int cache = plp->from;
			plp->from = plp->to;
			plp->to = cache;
		}

		printf("%d planes from %d to %d\n", plp->numbers, plp->from, plp->to);
		return 0;
	}

	// all other invalid inputs
	else {
		printf("Invalid command. Please check how to make valid command by '-h' or '--help'.\n");
		return 1;
	}
}


void read(float* dataArray, const char* fileName) {
	FILE* dataFile = fopen(fileName, "r");
	if (dataFile == NULL) {
		printf("Unable to open file: %s.\n", fileName);
		exit(1);
	}

	char line[500];
	int count = 0;
	// loop for each line
	while (fgets(line, sizeof(line), dataFile)) {
		char* token;
		token = strtok(line, ",");

		// check if this is an empty line
		if (strcmp(token, "\n") == 0) {
			printf("Finish reading file: %s at line %d.\n", fileName, count / 3 + 1);
			return;
		}

		// read 3 tokens of that line
		if (token == NULL) {
			printf("Can't read csv file properly on line %d.\n", count / 3 + 1);
			exit(1);
		}
		// write float of that token into dataArray
		dataArray[count] = (float)atof(token);
		++count;

		token = strtok(NULL, ",");
		if (token == NULL) {
			printf("Can't read csv file properly on line %d.\n", count / 3 + 1);
			exit(1);
		}
		dataArray[count] = (float)atof(token);
		++count;

		token = strtok(NULL, "\n");
		if (token == NULL) {
			printf("Can't read csv file properly on line %d.\n", count / 3 + 1);
			exit(1);
		}
		dataArray[count] = (float)atof(token);
		++count;

	}
	fclose(dataFile);
}


__device__
void matrixMul(float* matrix1, float* matrix2, float* result, int x, int y, int z) {
	for (int i = 0; i < x; ++i) {
		for (int j = 0; j < z; ++j) {
			float summ = 0;
			for (int k = 0; k < y; ++k) {
				summ += matrix1[i * y + k] * matrix2[k * z + j];
			}
			result[i * z + j] = summ;
		}
	}
}


__global__
void psalgo(int from, int to, int numbers, float* data3_d, float* data4_d, float* data5_d, float* data6_d, float* data7_d, float* result) {
	unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;
	if (tx < RESOLUTIONX && ty < RESOLUTIONY) {
		int planeCount = 0;
		float depth = from;
		float step = (float)(from - to) / (float)numbers;
		float wldCord[4];
		float pixCord[4];
		float projCord[3];
		float pixColor[15];
		int x, y;

		float miniResult = from;
		float miniLoss = -1;
    
		while (planeCount <= numbers) {
			
			pixCord[0] = tx * depth;
			pixCord[1] = ty * depth;
			pixCord[2] = depth;
			pixCord[3] = 1;
			matrixMul(pi5_d, pixCord, wldCord, 3, 4, 1);
			wldCord[3] = 1;
			
			// Projection on data3
			matrixMul(pm3_d, wldCord, projCord, 3, 4, 1);
			projCord[0] = projCord[0] / projCord[2];
			projCord[1] = projCord[1] / projCord[2];
			x = (int)round(projCord[0]);
			y = (int)round(projCord[1]);
			if (x >= RESOLUTIONX || x < 0 || y < 0 || y >= RESOLUTIONY) {
				pixColor[0] = -1;
				pixColor[1] = -1;
				pixColor[2] = -1;
			}
			else {
				int index = 3 * (y * RESOLUTIONX + x);
				pixColor[0] = data3_d[index];
				pixColor[1] = data3_d[index + 1];
				pixColor[2] = data3_d[index + 2];
			}
			
			// Projection on data4
			matrixMul(pm4_d, wldCord, projCord, 3, 4, 1);
			projCord[0] = projCord[0] / projCord[2];
			projCord[1] = projCord[1] / projCord[2];
			x = (int)round(projCord[0]);
			y = (int)round(projCord[1]);
			if (x >= RESOLUTIONX || x < 0 || y < 0 || y >= RESOLUTIONY) {
				pixColor[3] = -1;
				pixColor[4] = -1;
				pixColor[5] = -1;
			}
			else {
				int index = 3 * (y * RESOLUTIONX + x);
				pixColor[3] = data4_d[index];
				pixColor[4] = data4_d[index + 1];
				pixColor[5] = data4_d[index + 2];
			}
			// Projection on data5
			matrixMul(pm5_d, wldCord, projCord, 3, 4, 1);
			projCord[0] = projCord[0] / projCord[2];
			projCord[1] = projCord[1] / projCord[2];
			x = (int)round(projCord[0]);
			y = (int)round(projCord[1]);
			if (x >= RESOLUTIONX || x < 0 || y < 0 || y >= RESOLUTIONY) {
				pixColor[6] = -1;
				pixColor[7] = -1;
				pixColor[8] = -1;
			}
			else {
				int index = 3 * (y * RESOLUTIONX + x);
				pixColor[6] = data5_d[index];
				pixColor[7] = data5_d[index + 1];
				pixColor[8] = data5_d[index + 2];
			}
			// Projection on data6
			matrixMul(pm6_d, wldCord, projCord, 3, 4, 1);
			projCord[0] = projCord[0] / projCord[2];
			projCord[1] = projCord[1] / projCord[2];
			x = (int)round(projCord[0]);
			y = (int)round(projCord[1]);
			if (x >= RESOLUTIONX || x < 0 || y < 0 || y >= RESOLUTIONY) {
				pixColor[9] = -1;
				pixColor[10] = -1;
				pixColor[11] = -1;
			}
			else {
				int index = 3 * (y * RESOLUTIONX + x);
				pixColor[9] = data6_d[index];
				pixColor[10] = data6_d[index + 1];
				pixColor[11] = data6_d[index + 2];
			}
			// Projection on data7
			matrixMul(pm7_d, wldCord, projCord, 3, 4, 1);
			projCord[0] = projCord[0] / projCord[2];
			projCord[1] = projCord[1] / projCord[2];
			x = (int)round(projCord[0]);
			y = (int)round(projCord[1]);
			if (x >= RESOLUTIONX || x < 0 || y < 0 || y >= RESOLUTIONY) {
				pixColor[12] = -1;
				pixColor[13] = -1;
				pixColor[14] = -1;
			}
			else {
				int index = 3 * (y * RESOLUTIONX + x);
				pixColor[12] = data7_d[index];
				pixColor[13] = data7_d[index + 1];
				pixColor[14] = data7_d[index + 2];
			}
			
			
			// Now Calculate SAD
			float r = 0, g = 0, b = 0;
			int count = 0;
			for (int i = 0; i < 5; ++i) {
				if (pixColor[3 * i + 0] < 0) continue;
				r += pixColor[3 * i + 0];
				g += pixColor[3 * i + 1];
				b += pixColor[3 * i + 2];
				++count;
			}
			
			if (count > 2){
				r /= count; g /= count; b /= count;
				
				float loss = 0;
				for (int i = 0; i < 5; ++i) {
					if (pixColor[3 * i + 0] < 0) continue;
					loss += (float)fabs(pixColor[3 * i + 0] - r);
					loss += (float)fabs(pixColor[3 * i + 1] - g);
					loss += (float)fabs(pixColor[3 * i + 2] - b);
				}
				loss /= count;

				if (miniLoss < 0 || loss < miniLoss) {
					miniLoss = loss;
					miniResult = depth;
				}
				
			}
			depth -= step;
			++planeCount;
		}

		result[ty * RESOLUTIONX + tx] = miniResult;
		
	}
}




int main(int argc, char* argv[]) {
	clock_t start, finish;
	int total_time;

    float pm3[] = {1275.26, -2877.31, -148.52, 754.647, -747.178, -1000.76, 2663.54, -12946.3, -0.604314, -0.791759, -0.0890082, -10.1165};
    float pm4[] = {1768.22, -2606.58, -79.9353, 12002.6, -515.384, -1020.13, 2710.72, -10582.4, -0.453793, -0.889721, -0.0496901, -9.01598};
    float pm5[] = {2246.17, -2208.64, -62.134, 24477.4, -316.161, -1091.08, 2713.64, -8334.18, -0.269944, -0.961723, -0.0471142, -7.01217};
    float pm6[] = {2592.44, -1790, -48.8682, 35535.5, -114.072, -1095.02, 2728.03, -5423.57, -0.100616, -0.994335, -0.0342684, -4.72891};
    float pm7[] = {2890.6, -1253.12, -37.3055, 46750.1, 105.251, -1060.63, 2741.94, -1799.29, 0.0943235, -0.995311, -0.0214366, -1.68246};

	float pi5[] = { 0.00034888542585823415, -5.808190865675488e-06, -0.7946427612701072, -14.1604, -9.798911120446392e-05, -1.6072973979090537e-05, -0.7965288644222046, -3.32084, 1.2491902020089363e-06, 0.00036136950280672903, -0.41284422473450133, 0.0862032 };


    struct planeParam plp;
    int UIStatus;

    // UI
    UIStatus = UI(argc, argv, &plp);
    if (UIStatus != 0) {
        printf("\nApplication terminates.\n");
        return 0;
		}
	
	
    // Read png data into float array
    int dataSize = RESOLUTIONX * RESOLUTIONY * 3 * sizeof(float);
    float* data3 = (float*)malloc(dataSize);
    float* data4 = (float*)malloc(dataSize);
    float* data5 = (float*)malloc(dataSize);
    float* data6 = (float*)malloc(dataSize);
    float* data7 = (float*)malloc(dataSize);
    read(data3, "0003.csv");
    read(data4, "0004.csv");
    read(data5, "0005.csv");
    read(data6, "0006.csv");
    read(data7, "0007.csv");
    printf("Done reading pixels into array.\n");

    // allocate global memory on gpu
    float *data3_d, *data4_d, *data5_d, *data6_d, *data7_d, *result_d;
    checkCUDAError(hipMalloc((float**)&data3_d, dataSize));
    checkCUDAError(hipMalloc((float**)&data4_d, dataSize));
    checkCUDAError(hipMalloc((float**)&data5_d, dataSize));
    checkCUDAError(hipMalloc((float**)&data6_d, dataSize));
    checkCUDAError(hipMalloc((float**)&data7_d, dataSize));
	checkCUDAError(hipMalloc((float**)&result_d, RESOLUTIONX * RESOLUTIONY * sizeof(float)));

    // write png data into gpu
    checkCUDAError(hipMemcpy(data3_d, data3, dataSize, hipMemcpyHostToDevice));
    checkCUDAError(hipMemcpy(data4_d, data4, dataSize, hipMemcpyHostToDevice));
    checkCUDAError(hipMemcpy(data5_d, data5, dataSize, hipMemcpyHostToDevice));
    checkCUDAError(hipMemcpy(data6_d, data6, dataSize, hipMemcpyHostToDevice));
    checkCUDAError(hipMemcpy(data7_d, data7, dataSize, hipMemcpyHostToDevice));

	checkCUDAError(hipMemcpyToSymbol(HIP_SYMBOL(pm3_d), pm3, 12 * sizeof(float)));
	checkCUDAError(hipMemcpyToSymbol(HIP_SYMBOL(pm4_d), pm4, 12 * sizeof(float)));
	checkCUDAError(hipMemcpyToSymbol(HIP_SYMBOL(pm5_d), pm5, 12 * sizeof(float)));
	checkCUDAError(hipMemcpyToSymbol(HIP_SYMBOL(pm6_d), pm6, 12 * sizeof(float)));
	checkCUDAError(hipMemcpyToSymbol(HIP_SYMBOL(pm7_d), pm7, 12 * sizeof(float)));

	checkCUDAError(hipMemcpyToSymbol(HIP_SYMBOL(pi5_d), pi5, 12 * sizeof(float)));
	
	
	// defining dim and grid, then launch kernel
	dim3 threads(16, 16);
	dim3 grid((int)ceil(1.0 * RESOLUTIONX / threads.x), (int)ceil(1.0 * RESOLUTIONY / threads.y));
	start = clock();
	printf("Now launching kernel.\n");
	psalgo<<<grid, threads>>>(plp.from, plp.to, plp.numbers, data3_d, data4_d, data5_d, data6_d, data7_d, result_d);
	hipError_t error_check = hipGetLastError();
	if (error_check != hipSuccess) {
		printf("%s\n", hipGetErrorString(error_check));
		return 0;
	}
	checkCUDAError(hipDeviceSynchronize());
	finish = clock();
	total_time = (int)(finish - start);
	printf("\nDone psalgo with GPU in %d miliseconds.\n", total_time);

	// get result from gpu
	float* result = (float*)malloc(RESOLUTIONX * RESOLUTIONY * sizeof(float));
	checkCUDAError(hipMemcpy(result, result_d, RESOLUTIONX * RESOLUTIONY * sizeof(float), hipMemcpyDeviceToHost));

	// write result into csv
	FILE* output = fopen("output.csv", "w");
	if (output == NULL) {
		printf("Can't open file for output.\n");
		return 1;
	}
	for (int i = 0; i < RESOLUTIONX * RESOLUTIONY; ++i) {
		fprintf(output, "%f\n", result[i]);
	}
	fclose(output);
	
	// free cuda memory
	checkCUDAError(hipFree(data3_d));
	checkCUDAError(hipFree(data4_d));
	checkCUDAError(hipFree(data5_d));
	checkCUDAError(hipFree(data6_d));
	checkCUDAError(hipFree(data7_d));
	checkCUDAError(hipFree(result_d));

}			 

